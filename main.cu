// clang-format off
#include <glad/glad.h>
#include <GLFW/glfw3.h>
// clang-format on

#include <iomanip>
#include <iostream>
#include <thread>
#include <vector>

class Shader {
public:
    Shader() : ID(0) {}

    // constructor generates the shader on the fly
    // ------------------------------------------------------------------------
    void build() {
        std::string vertex_code_str = "#version 330 core\n"
                                      "layout (location = 0) in vec3 aPos;\n"
                                      "layout (location = 1) in vec3 aColor;\n"
                                      "layout (location = 2) in vec2 aTexCoord;\n"
                                      "\n"
                                      "out vec3 ourColor;\n"
                                      "out vec2 TexCoord;\n"
                                      "\n"
                                      "void main()\n"
                                      "{\n"
                                      "    gl_Position = vec4(aPos, 1.0);\n"
                                      "    ourColor = aColor;\n"
                                      "    TexCoord = vec2(aTexCoord.x, aTexCoord.y);\n"
                                      "}\n";

        std::string fragment_code_str = "#version 330 core\n"
                                        "out vec4 FragColor;\n"
                                        "\n"
                                        "in vec3 ourColor;\n"
                                        "in vec2 TexCoord;\n"
                                        "\n"
                                        "// texture sampler\n"
                                        "uniform sampler2D texture1;\n"
                                        "\n"
                                        "void main()\n"
                                        "{\n"
                                        "    FragColor = texture(texture1, TexCoord);\n"
                                        "}\n";

        const char *vShaderCode = vertex_code_str.c_str();
        const char *fShaderCode = fragment_code_str.c_str();
        // 2. compile shaders
        unsigned int vertex, fragment;
        // vertex shader
        vertex = glCreateShader(GL_VERTEX_SHADER);
        glShaderSource(vertex, 1, &vShaderCode, NULL);
        glCompileShader(vertex);
        checkCompileErrors(vertex, "VERTEX");
        // fragment Shader
        fragment = glCreateShader(GL_FRAGMENT_SHADER);
        glShaderSource(fragment, 1, &fShaderCode, NULL);
        glCompileShader(fragment);
        checkCompileErrors(fragment, "FRAGMENT");
        // shader Program
        ID = glCreateProgram();
        glAttachShader(ID, vertex);
        glAttachShader(ID, fragment);
        glLinkProgram(ID);
        checkCompileErrors(ID, "PROGRAM");
        // delete the shaders as they're linked into our program now and no longer necessary
        glDeleteShader(vertex);
        glDeleteShader(fragment);
    }

    // activate the shader
    // ------------------------------------------------------------------------
    void use() { glUseProgram(ID); }

private:
    unsigned int ID;

    // utility function for checking shader compilation/linking errors.
    // ------------------------------------------------------------------------
    void checkCompileErrors(unsigned int shader, std::string type) {
        int success;
        char infoLog[1024];
        if (type != "PROGRAM") {
            glGetShaderiv(shader, GL_COMPILE_STATUS, &success);
            if (!success) {
                glGetShaderInfoLog(shader, 1024, NULL, infoLog);
                std::cout << "ERROR::SHADER_COMPILATION_ERROR of type: " << type << "\n"
                          << infoLog << "\n -- --------------------------------------------------- -- " << std::endl;
            }
        }
        else {
            glGetProgramiv(shader, GL_LINK_STATUS, &success);
            if (!success) {
                glGetProgramInfoLog(shader, 1024, NULL, infoLog);
                std::cout << "ERROR::PROGRAM_LINKING_ERROR of type: " << type << "\n"
                          << infoLog << "\n -- --------------------------------------------------- -- " << std::endl;
            }
        }
    }
};


class GLHelper {
    uint VBO = 0;
    uint VAO = 0;
    uint EBO = 0;
    GLFWwindow *window = nullptr;

    Shader shader;
    unsigned int texture = 0;

    bool initialized = false;

    int resolution_x = 0;
    int resolution_y = 0;

public:
    uint8_t *gpu_frame_buffer = nullptr;
    // std::vector<uint8_t> frame_buffer;

    ~GLHelper() {
        if (initialized) {
            this->release();
        }
    }

    void init(const std::string &title, int width, int height) {
        initialized = true;

        resolution_x = width;
        resolution_y = height;

        const uint num_pixels = width * height;

        /*
        frame_buffer = std::vector<uint8_t>(num_pixels * 3);
        for (uint idx = 0; idx < num_pixels; ++idx) {
            frame_buffer[idx * 3 + 0] = 0;
            frame_buffer[idx * 3 + 1] = 0;
            frame_buffer[idx * 3 + 2] = 0;
        }
        */


        const auto size = sizeof(uint8_t) * 3 * num_pixels;
        hipMallocManaged(&gpu_frame_buffer, size);
        for (uint idx = 0; idx < num_pixels; ++idx) {
            gpu_frame_buffer[idx * 3 + 0] = 0;
            gpu_frame_buffer[idx * 3 + 1] = 0;
            gpu_frame_buffer[idx * 3 + 2] = 0;
        }


        glfwInit();
        glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
        glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
        glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
        glfwWindowHint(GLFW_RESIZABLE, GLFW_FALSE);
        // thus disable window resizing

        create_window(width, height, title);
        glfwMakeContextCurrent(window);

        /*
        // center the window
        glfwSetWindowPos(window, (monitor_resolution.x - window_dimension.x) / 2,
                         (monitor_resolution.y - window_dimension.y) / 2);
        */

        // glad: load all OpenGL function pointers
        // ---------------------------------------
        if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress)) {
            std::cout << "ERROR: failed to initialize GLAD\n";
            exit(1);
        }

        build_triangles();
    }

    void release() {
        // optional: de-allocate all resources once they've outlived their purpose:
        // ------------------------------------------------------------------------
        glDeleteVertexArrays(1, &VAO);
        glDeleteBuffers(1, &VBO);
        glDeleteBuffers(1, &EBO);

        glfwTerminate();
    }

    void create_window(uint width, uint height, const std::string &window_initial_name) {
        window = glfwCreateWindow(width, height, window_initial_name.c_str(), NULL, NULL);
        if (window == NULL) {
            std::cout << "ERROR: failed to create GLFW window" << std::endl;
            glfwTerminate();
            exit(1);
        }
    }

    /*
    static std::string assemble_title(const float progress_percentage) {
        std::stringstream stream;
        stream << std::fixed << std::setprecision(1) << (progress_percentage * 100.0);
        return stream.str() + "%";
    }
    */

    void draw_frame(const std::string &title) {
        /*
        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, resolution_x, resolution_y, 0, GL_RGB, GL_UNSIGNED_BYTE,
                     this->frame_buffer.data());
        */

        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, resolution_x, resolution_y, 0, GL_RGB, GL_UNSIGNED_BYTE,
                     this->gpu_frame_buffer);

        glGenerateMipmap(GL_TEXTURE_2D);
        glBindTexture(GL_TEXTURE_2D, texture);

        shader.use();
        glBindVertexArray(VAO);
        glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);
        glfwSwapBuffers(window);

        glfwSetWindowTitle(window, title.c_str());
        glfwPollEvents();
    }

    void build_triangles() {
        shader.build();

        // set up vertex data (and buffer(s)) and configure vertex attributes
        // ------------------------------------------------------------------
        const float vertices[] = {
                // positions          // colors           // texture coords
                1.0f,  1.0f,  0.0f, 1.0f, 1.0f, 1.0f, 1.0f, 0.0f, // top right
                1.0f,  -1.0f, 0.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, // bottom right
                -1.0f, -1.0f, 0.0f, 1.0f, 1.0f, 1.0f, 0.0f, 1.0f, // bottom left
                -1.0f, 1.0f,  0.0f, 1.0f, 1.0f, 1.0f, 0.0f, 0.0f // top left
        };

        const unsigned int indices[] = {
                0, 1, 3, // first triangle
                1, 2, 3 // second triangle
        };

        glGenVertexArrays(1, &VAO);
        glGenBuffers(1, &VBO);
        glGenBuffers(1, &EBO);

        glBindVertexArray(VAO);

        glBindBuffer(GL_ARRAY_BUFFER, VBO);
        glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

        glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO);
        glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices), indices, GL_STATIC_DRAW);

        // position attribute
        glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void *)0);
        glEnableVertexAttribArray(0);
        // color attribute
        glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void *)(3 * sizeof(float)));
        glEnableVertexAttribArray(1);
        // texture coord attribute
        glVertexAttribPointer(2, 2, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void *)(6 * sizeof(float)));
        glEnableVertexAttribArray(2);

        // load and create a texture
        // -------------------------
        glGenTextures(1, &texture);
        glBindTexture(GL_TEXTURE_2D,
                      texture); // all upcoming GL_TEXTURE_2D operations now have effect on this
        // texture object
        // set the texture wrapping parameters
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S,
                        GL_REPEAT); // set texture wrapping to GL_REPEAT (default wrapping method)
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
        // set texture filtering parameters
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR_MIPMAP_LINEAR);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    }
};


int main() {
    GLHelper gl_helper;

    /*
     * bug: (755, 1200)
     * weird result: (750, 1200)
     */

    int width = 755;
    int height = 1200;

    gl_helper.init("initializing", width, height);

    auto title = "resolution: " + std::to_string(width) + "x" + std::to_string(height);
    gl_helper.draw_frame(title);

    getchar();

    return 0;
}
